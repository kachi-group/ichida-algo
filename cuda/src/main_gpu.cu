#include "hip/hip_runtime.h"
#include "../include/matrix.h"
#include <dirent.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define NUM_LAYERS 7

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t err = call;                                                                                        \
        if (err != hipSuccess) {                                                                                      \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n", __func__, __FILE__, __LINE__, hipGetErrorString(err));  \
            exit(EXIT_FAILURE);                                                                                        \
        }                                                                                                              \
    } while (0)

matrix* weights[NUM_LAYERS];
matrix* biases[NUM_LAYERS];

// device weights and biases;
matrix* d_weights;
matrix* d_biases;
matrix* d_input;
// allocating device matrix for weights and biases
// CUDA_CHECK(hipMalloc(&d_weights, NUM_LAYERS * sizeof(matrix)));
// CUDA_CHECK(hipMalloc(&d_biases, NUM_LAYERS * sizeof(matrix)));

char letters[52] = {'A', 'a', 'B', 'b', 'C', 'c', 'D', 'd', 'E', 'e', 'F', 'f', 'G', 'g', 'H', 'h', 'I', 'i',
                    'J', 'j', 'K', 'k', 'L', 'l', 'M', 'm', 'N', 'n', 'O', 'o', 'P', 'p', 'Q', 'q', 'R', 'r',
                    'S', 's', 'T', 't', 'U', 'u', 'V', 'v', 'W', 'w', 'X', 'x', 'Y', 'y', 'Z', 'z'};

void process_weights_str(char* line, int layer) {
    char* token;
    float value;
    const char* delimiter = ",";

    token = strtok(line, delimiter);
    int n = (weights[layer]->rows) * (weights[layer]->cols);
    for (int i = 0; i < n; i++) {
        value = strtof(token, NULL);
        (weights[layer]->data)[i] = value;
        token = strtok(NULL, delimiter);
    }
}

void process_biases_str(char* line, int layer) {
    char* token;
    float value;
    const char* delimiter = ",";

    token = strtok(line, delimiter);

    int n = biases[layer]->rows;
    for (int i = 0; i < n; i++) {
        value = strtof(token, NULL);
        (biases[layer]->data)[i] = value;
        token = strtok(NULL, delimiter);
    }
}

void read_model(const char* file_name) {
    FILE* file = fopen(file_name, "r");

    char* line = NULL;
    size_t len = 0;
    int line_number = 0;
    int layer = 0;

    while ((getline(&line, &len, file)) != -1) {
        if ((line_number - 1) % 4 == 0) {
            process_weights_str(line, layer);
        } else if ((line_number - 3) % 4 == 0) {
            process_biases_str(line, layer);
            layer++;
        }
        line_number++;
    }

    free(line);
    fclose(file);
}

void read_tensor(matrix* a, const char* fileName) {
    FILE* file = fopen(fileName, "r");
    char* line = NULL;
    size_t len = 0;

    getline(&line, &len, file);
    char* token;
    float value;
    const char* delimiter = ",";
    token = strtok(line, delimiter);

    for (int i = 0; i < 225; i++) {
        value = strtof(token, NULL);
        (a->data)[i] = value;
        token = strtok(NULL, delimiter);
    }
    free(line);
    fclose(file);
}

void propagate_fwd(matrix* weights, matrix* input_layer, matrix* output_layer, matrix* biases) {
    // everything here is device code
    // matrix_mul(weights, input_layer, output_layer);
    matrix_mul<<<1, 1>>>(weights, input_layer, output_layer);
    hipDeviceSynchronize();
    matrix_add<<<1, 1>>>(output_layer, biases);
    hipDeviceSynchronize();
}

// Get result from output layer
__global__ void get_max(matrix* a, int* d_int) {
    int idx = 0;
    float res = a->data[0];
    for (int i = 0; i < a->rows; i++) {
        if (res < a->data[i]) {
            res = a->data[i];
            idx = i;
        }
    }
    *d_int = idx;
}

int infer(matrix* d_input) {
    matrix* mdl_layers[NUM_LAYERS];    // host
    matrix* d_mdl_layers;              // device
    mdl_layers[0] = new_matrix(98, 1); // you may see garbage values as it is unitialized
    mdl_layers[1] = new_matrix(65, 1);
    mdl_layers[2] = new_matrix(50, 1);
    mdl_layers[3] = new_matrix(30, 1);
    mdl_layers[4] = new_matrix(25, 1);
    mdl_layers[5] = new_matrix(40, 1);
    mdl_layers[6] = new_matrix(52, 1);

    CUDA_CHECK(hipMalloc(&d_mdl_layers, NUM_LAYERS * sizeof(matrix)));
    initmalloc(&d_mdl_layers[0], mdl_layers[0], 98, 1);
    initmalloc(&d_mdl_layers[1], mdl_layers[1], 65, 1);
    initmalloc(&d_mdl_layers[2], mdl_layers[2], 50, 1);
    initmalloc(&d_mdl_layers[3], mdl_layers[3], 30, 1);
    initmalloc(&d_mdl_layers[4], mdl_layers[4], 25, 1);
    initmalloc(&d_mdl_layers[5], mdl_layers[5], 40, 1);
    initmalloc(&d_mdl_layers[6], mdl_layers[6], 52, 1);

    propagate_fwd(&d_weights[0], d_input, &d_mdl_layers[0], &d_biases[0]);
    relu<<<1, 1>>>(&d_mdl_layers[0]);
    hipDeviceSynchronize();

    propagate_fwd(&d_weights[1], &d_mdl_layers[0], &d_mdl_layers[1], &d_biases[1]);
    relu<<<1, 1>>>(&d_mdl_layers[1]);
    hipDeviceSynchronize();

    propagate_fwd(&d_weights[2], &d_mdl_layers[1], &d_mdl_layers[2], &d_biases[2]);
    relu<<<1, 1>>>(&d_mdl_layers[2]);
    hipDeviceSynchronize();

    propagate_fwd(&d_weights[3], &d_mdl_layers[2], &d_mdl_layers[3], &d_biases[3]);
    relu<<<1, 1>>>(&d_mdl_layers[3]);
    hipDeviceSynchronize();

    propagate_fwd(&d_weights[4], &d_mdl_layers[3], &d_mdl_layers[4], &d_biases[4]);
    relu<<<1, 1>>>(&d_mdl_layers[4]);
    hipDeviceSynchronize();

    propagate_fwd(&d_weights[5], &d_mdl_layers[4], &d_mdl_layers[5], &d_biases[5]);
    relu<<<1, 1>>>(&d_mdl_layers[5]);
    hipDeviceSynchronize();

    propagate_fwd(&d_weights[6], &d_mdl_layers[5], &d_mdl_layers[6], &d_biases[6]);
    softmax<<<1, 1>>>(&d_mdl_layers[6]);
    hipDeviceSynchronize();

    int* d_int;
    int h_int = 0;

    CUDA_CHECK(hipMalloc((void**)&d_int, sizeof(int)));
    get_max<<<1, 1>>>(&d_mdl_layers[6], d_int);
    hipDeviceSynchronize();
    CUDA_CHECK(hipMemcpy(&h_int, d_int, sizeof(int), hipMemcpyDeviceToHost));

    dealloc(&d_mdl_layers[0]);
    dealloc(&d_mdl_layers[1]);
    dealloc(&d_mdl_layers[2]);
    dealloc(&d_mdl_layers[3]);
    dealloc(&d_mdl_layers[4]);
    dealloc(&d_mdl_layers[5]);
    dealloc(&d_mdl_layers[6]);

    return h_int;
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        printf("Not enough arguments.");
        return EXIT_FAILURE;
    }

    // Start timing
    struct timeval stop, start;
    gettimeofday(&start, NULL);

    // TODO: find a way to load static weights and biases
    // Load model (The memory of those code should be initialize during compile time to enchance the speed)
    weights[0] = new_matrix(98, 225);
    weights[1] = new_matrix(65, 98);
    weights[2] = new_matrix(50, 65);
    weights[3] = new_matrix(30, 50);
    weights[4] = new_matrix(25, 30);
    weights[5] = new_matrix(40, 25);
    weights[6] = new_matrix(52, 40);

    biases[0] = new_matrix(98, 1);
    biases[1] = new_matrix(65, 1);
    biases[2] = new_matrix(50, 1);
    biases[3] = new_matrix(30, 1);
    biases[4] = new_matrix(25, 1);
    biases[5] = new_matrix(40, 1);
    biases[6] = new_matrix(52, 1);

    read_model(argv[1]);
    // initialize d_weights struct matrix arr;
    CUDA_CHECK(hipMalloc(&d_weights, NUM_LAYERS * sizeof(matrix)));
    CUDA_CHECK(hipMalloc(&d_biases, NUM_LAYERS * sizeof(matrix)));
    initmalloc(&d_weights[0], weights[0], 98, 225);
    initmalloc(&d_weights[1], weights[1], 65, 98);
    initmalloc(&d_weights[2], weights[2], 50, 65);
    initmalloc(&d_weights[3], weights[3], 30, 50);
    initmalloc(&d_weights[4], weights[4], 25, 30);
    initmalloc(&d_weights[5], weights[5], 40, 25);
    initmalloc(&d_weights[6], weights[6], 52, 40);
    initmalloc(&d_biases[0], biases[0], 98, 1);
    initmalloc(&d_biases[1], biases[1], 65, 1);
    initmalloc(&d_biases[2], biases[2], 50, 1);
    initmalloc(&d_biases[3], biases[3], 30, 1);
    initmalloc(&d_biases[4], biases[4], 25, 1);
    initmalloc(&d_biases[5], biases[5], 40, 1);
    initmalloc(&d_biases[6], biases[6], 52, 1);

    // Run program
    const char* directory_path = argv[2];
    struct dirent* entry;
    DIR* dir = opendir(directory_path);

    // Read and process inputs
    char* file_name = (char*)malloc((100) * sizeof(char));
    char* file_num_str = (char*)malloc((100) * sizeof(char));

    int file_num;
    int size = 0;
    while ((entry = readdir(dir)) != NULL) {
        if (entry->d_type == DT_REG) {
            size++;
        }
    }
    int* results = (int*)malloc((size + 1) * sizeof(int));
    dir = opendir(directory_path);
    matrix* d_input;

    while ((entry = readdir(dir)) != NULL) {
        if (entry->d_type == DT_REG) {
            matrix* input = new_matrix(225, 1);
            strcpy(file_num_str, entry->d_name);
            file_num_str[strlen(entry->d_name) - 7] = '\0';
            file_num = atoi(entry->d_name);
            strcpy(file_name, directory_path);
            strcat(file_name, "/");
            strcat(file_name, entry->d_name);
            read_tensor(input, file_name);
            CUDA_CHECK(hipMalloc(&d_input, 255 * sizeof(matrix)));
            initmalloc(d_input, input, 1, 225);
            results[file_num] = infer(d_input);
            dealloc(d_input);

            free(input);
        }
    }

    free(file_name);
    free(file_num_str);
    closedir(dir);

    // Write to csv file
    FILE* csv_file = fopen("results.csv", "w+");
    fprintf(csv_file, "image_number, guess\n");
    for (int i = 1; i <= size; i++) {
        fprintf(csv_file, "%d, %c\n", i, letters[results[i]]);
    }
    fclose(csv_file);

    // Time taken
    gettimeofday(&stop, NULL);
    printf("took %lu us\n", (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec);

    return EXIT_SUCCESS;
}